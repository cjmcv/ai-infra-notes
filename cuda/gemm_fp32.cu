#include "hip/hip_runtime.h"
%%cuda
/*!
* \brief gemm: C = A * B.
*/
#include <iostream>
#include <hip/hip_runtime.h>
#include ""
#include "time.h"

////////////////
// Macro.
////////////////
#define CUDA_CHECK(condition) \
    do { \
        hipError_t error = condition; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA_CHECK error in line %d of file %s : %s \n", \
                    __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
            exit(EXIT_FAILURE); \
        } \
    } while(0);

////////////////
// Structure.
////////////////

// Timer for cuda.
struct GpuTimer {
    GpuTimer() {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
    }
    ~GpuTimer() {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }
    void Start() {
        hipEventRecord(start_, NULL);
    }
    void Stop() {
        hipEventRecord(stop_, NULL);
    }
    float ElapsedMillis() {
        float elapsed;
        hipEventSynchronize(stop_);
        hipEventElapsedTime(&elapsed, start_, stop_);
        return elapsed;
    }

    hipEvent_t start_;
    hipEvent_t stop_;
};

////////////////
// Function.
////////////////

// 
int InitEnvironment(const int dev_id) {
    CUDA_CHECK(hipSetDevice(dev_id));
    hipDeviceProp_t device_prop;
    CUDA_CHECK(hipGetDeviceProperties(&device_prop, dev_id));
    if (device_prop.computeMode == hipComputeModeProhibited) {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        return 1;
    }
    fprintf(stderr, "GPU Device %d: \"%s\" with compute capability %d.%d with %d multi-processors.\n\n", 
      dev_id, device_prop.name, device_prop.major, device_prop.minor, device_prop.multiProcessorCount);

    return 0;
}

void CleanUpEnvironment() {
    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    CUDA_CHECK(hipDeviceReset());
}

////////////////////////////////////////////////////////////////////////////////

// Initialize the input data.
void GenMatrix(const int height, const int width, float *mat) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            mat[i*width + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX*RAND_MAX);
        }
    }
}

// Just for checking the result.
float GetMean(const float* mat, const int height, const int width) {
    int num = height * width;
    float total = 0;
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            total += mat[i*width + j];
        }
    }
    return total / num;
}

// Just for checking the result too.
void MatrixPrint(const float* mat, const int height, const int width) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            std::cout << mat[i*width + j] << ",";
        }
        std::cout << std::endl;
    }
}

// CPU version 1: 1583 ms
// 普通实现版本
void GemmHostV1(const int M, const int N, const int K,
    const float *A, const int lda,
    const float *B, const int ldb,
    float *C, const int ldc) {
    int i, j, k;
    memset(C, 0, sizeof(float) * ldc * M);
    for (i = 0; i < M; ++i) {
        for (j = 0; j < N; ++j) {
            for (k = 0; k < K; ++k) {
                C[i*ldc + j] += A[i*lda + k]*B[k*ldb + j];
            }
        }
    }
}

// CPU version 2: 3389 ms
// 按i和j方向分块的矩阵乘法，便于改写成cuda
// （暂时省略边界处理）
void GemmHostV2(const int M, const int N, const int K,
                const float *A, const int lda,
                const float *B, const int ldb,
                float *C, const int ldc) {
    int bi, bj;
    int i, j, k;
    const int block_size = 32;
    int block_num_M = M / block_size;
    int block_num_N = N / block_size;
    memset(C, 0, sizeof(float) * ldc * M);

    // Loop over all of the blocks.
    for (bi = 0; bi < block_num_M; ++bi) {
        for (bj = 0; bj < block_num_N; ++bj) {
            // Loop over all of the elements in a block.
            for (i = bi*block_size; i < (bi + 1)*block_size; ++i) {
                for (j = bj*block_size; j < (bj + 1)*block_size; ++j) { 
                    for (k = 0; k < K; ++k) {
                        C[i*ldc + j] += A[i*lda + k] * B[k*ldb + j];
                    }
                }
            }
        }
    }
}

// CUDA version 1: 72 ms、
// 基于GemmHostV2直接一一对应改写而成,
// 其中的 bi,bj 使用 blockIdx.x,blockIdx.y 代替
// 其中的 i,j 使用 threadIdx.x,threadIdx.y 代替
// (注意：如GemmHostV2中block应为正方形)
// 所以去掉块内线程i/j和块的bi/bj，只需留下 k 循环.
//
// \ C[ty, tx] = A[ty, k] * B[k, tx]
// for k -> K
//     C[bi*bs + ty, bj*bs + tx] += A[bi*bs + ty, k] * B[k, bj*bs + tx]
__global__ void GemmKernelv1(const int M, const int N, const int K,
                             const float *A, const int lda,
                             const float *B, const int ldb,
                             float *C, const int ldc) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    float c_sub_acc = 0;
    for (int k = 0; k < K; k++) {
        c_sub_acc += A[i * lda + k] * B[k * ldb + j];
    }
    C[i * ldc + j] = c_sub_acc;
}

// CUDA version 2.
// 使用共享内存优化：先将数据从全局内存拷贝到共享内存，在共享内存中进行乘加运算，最后写回全局内存
//    因为共享内存以block划分，所以需要将逐个block的数据填充到shared[threadIdx.y][threadIdx.x]中，
// 则A和B矩阵均往各自K方向取block的数据进行填充。所以k方向多拆一个循环来索引块。
// 最终从多次读取全局内存计算 变成 一次读取全局内存到共享内存，多次读取共享内存计算
// 参考host端三层循环，对于最内层循环，A读取会重复 j 次，B读取会重复 i 次
// ps: 用template <int BLOCK_SIZE>的原因是kernel内以固定大小的方式开辟共享内存空间，无法使用变量blockDim
template <int BLOCK_SIZE>
__global__ void GemmKernelv2(const int M, const int N, const int K,
                             const float *A, const int lda,
                             const float *B, const int ldb,
                             float *C, const int ldc) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float a_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float b_shared[BLOCK_SIZE][BLOCK_SIZE];

    float c_sub_acc = 0;
    // 按 K 方向分块读入共享内存，一次读一个block
    for (int bk = 0; bk < K; bk += BLOCK_SIZE) {
        a_shared[threadIdx.y][threadIdx.x] = A[i * lda + (bk + threadIdx.x)];
        b_shared[threadIdx.y][threadIdx.x] = B[(bk + threadIdx.y) * ldb + j];
        // 等待块内线程同步
        __syncthreads();

        // 计算块内元素
        for (int k = 0; k < BLOCK_SIZE; k++) {
            c_sub_acc += a_shared[threadIdx.y][k] * b_shared[k][threadIdx.x];
        }
        // 再次同步，避免该块内个别线程已经计算完进入下一次循环中，往共享内存写数据，与正在共享内存正在计算中的数据相冲突
        __syncthreads();
    }

    C[i * ldc + j] += c_sub_acc;
}

// CUDA version 3.
//   分析v2，计算的过程实质为全局内存->共享内存->寄存器内存，则v2的k循环中需重复访问的数据存在于共享内存中。
// 就会有重复的从共享内存读取数据到寄存器。可考虑子在一次读取到共享内存后，再进分块一次读取到寄存器中，
// 使重复读取数据进行计算的操作放到更快的寄存器中完成。
template <int BLOCK_SIZE>
__global__ void GemmKernelv3(const int M, const int N, const int K,
                             const float *A, const int lda,
                             const float *B, const int ldb,
                             float *C, const int ldc) {
    
    int gid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int gid_y = blockIdx.y * blockDim.y + threadIdx.y;

    const int STEP = 2;
    float a_reg[STEP] = {0};
    float b_reg[STEP] = {0};
    float c_reg[STEP][STEP] = {{0}};
    __shared__ float a_shared[BLOCK_SIZE*STEP][BLOCK_SIZE*STEP];
    __shared__ float b_shared[BLOCK_SIZE*STEP][BLOCK_SIZE*STEP];

    int gid_sx = gid_x * STEP;
    int gid_sy = gid_y * STEP;    
    int tid_sx = threadIdx.x * STEP;
    int tid_sy = threadIdx.y * STEP;

    // 按 K 方向分块读入共享内存，一次读取临近的四个block, 一个线程处理四个元素
    for (int bk = 0; bk < K; bk += BLOCK_SIZE*STEP) {
        for (int si=0; si<STEP; si++) {
            for (int sj=0; sj<STEP; sj++) {
                a_shared[tid_sy+si][tid_sx+sj] = A[(gid_sy+si) * lda + (bk + tid_sx+sj)];
                b_shared[tid_sy+si][tid_sx+sj] = B[(bk + tid_sy+si) * ldb + gid_sx+sj];
            }
        }
        
        // 等待块内线程同步
        __syncthreads();

        // 计算块内元素, 每个线程处理临近四个元素
        // for (int k = 0; k < BLOCK_SIZE*STEP; k++) {
        //     for (int si=0; si<STEP; si++) {
        //         for (int sj=0; sj<STEP; sj++) {
        //             c_reg[si][sj] += a_shared[tid_sy+si][k] * b_shared[k][tid_sx+sj];
        //         }
        //     }
        // }
        for (int k = 0; k < BLOCK_SIZE*STEP; k++) {
            for (int s = 0; s < STEP; s++) {
                a_reg[s] = a_shared[tid_sy+s][k];
                b_reg[s] = b_shared[k][tid_sx+s];
            }
            for (int si=0; si<STEP; si++) {
                for (int sj=0; sj<STEP; sj++) {
                    c_reg[si][sj] += a_reg[si] * b_reg[sj];
                }
            }
        }

        // 再次同步，避免该块内个别线程已经计算完进入下一次循环中，往共享内存写数据，与正在共享内存正在计算中的数据相冲突
        __syncthreads();
    }

    for (int si=0; si<STEP; si++) {
        for (int sj=0; sj<STEP; sj++) {
            C[(gid_sy+si) * ldc + gid_sx+sj] += c_reg[si][sj];
            // printf("%f(%d, %d) \n", C[(gid_sy+si) * ldc + gid_sx+sj], (gid_sy+si), gid_sx+sj);
        }
    }
}

float MatrixMulCUDA(int version_id, const int M, const int N, const int K,
                    const float *A, const int lda,
                    const float *B, const int ldb,
                    float *C, const int ldc) {
    GpuTimer gpu_timer;

    const int block_side_size = 32;
    dim3 threads_per_block(block_side_size, block_side_size);
    dim3 blocks_per_grid((N + threads_per_block.x - 1) / threads_per_block.x, (M + threads_per_block.y - 1) / threads_per_block.y);
    
    // Warm up.
    for (int i=0; i<10; i++) {
        GemmKernelv1<< <blocks_per_grid, threads_per_block >> >
            (M, N, K, A, lda, B, ldb, C, ldc);        
    }
    hipMemset(C, 0, sizeof(float) * M * N);

    // Record the start event
    gpu_timer.Start();

    if (version_id == 1) {
        GemmKernelv1<< <blocks_per_grid, threads_per_block >> >
            (M, N, K, A, lda, B, ldb, C, ldc);        
    }
    else if (version_id == 2) {
        GemmKernelv2<block_side_size> << <blocks_per_grid, threads_per_block >> >
            (M, N, K, A, lda, B, ldb, C, ldc);    
    }
    else if (version_id == 3) {
        // 一个线程处理四个数据，一个block内线程数xy都减半，然后block的数量不变。
        const int step = 2;
        // const int block_side_size_new = block_side_size / step;
        // dim3 threads_per_block_r(block_side_size_new, block_side_size_new);
        // GemmKernelv3<block_side_size_new> << <blocks_per_grid, threads_per_block_r >> >
        //     (M, N, K, A, lda, B, ldb, C, ldc);

        dim3 blocks_per_grid_r(blocks_per_grid.x/step, blocks_per_grid.y/step);
        GemmKernelv3<block_side_size> << <blocks_per_grid_r, threads_per_block >> >
            (M, N, K, A, lda, B, ldb, C, ldc);    
    }

    // Record the stop event
    gpu_timer.Stop();

    return gpu_timer.ElapsedMillis();
}

#define TEST_CUDA_MODULE_UKERNEL(version_id)                                  \
    do {                                                                      \
        CUDA_CHECK(hipMemcpy(d_a, h_a, mem_size_a, hipMemcpyHostToDevice)); \
        CUDA_CHECK(hipMemcpy(d_b, h_b, mem_size_b, hipMemcpyHostToDevice)); \
        msec_total = MatrixMulCUDA(version_id, height_a, width_b, width_a, d_a, width_a, d_b, width_b, d_c, width_b); \
        CUDA_CHECK(hipMemcpy(h_c, d_c, mem_size_c, hipMemcpyDeviceToHost)); \
        printf("gpu version %d -> time: %f s, mean value = %f\n", version_id, msec_total/1000.f, GetMean(h_c, height_a, width_b)); \
    } while (0)

int main() {
    int ret = InitEnvironment(0);
    if (ret != 0) {
        printf("Failed to initialize the environment for cuda.");
        return -1;
    }

    int height_a = 1280, width_a = 4096;
    int height_b = 4096, width_b = 2048;
    if (width_a != height_b) {
        printf("width_a should be equal to height_b.\n");
        return 1;
    }

    const int mem_size_a = sizeof(float) * height_a * width_a;
    const int mem_size_b = sizeof(float) * height_b * width_b;
    const int mem_size_c = sizeof(float) * height_a * width_b;

    float *h_a = (float *)malloc(mem_size_a);
    float *h_b = (float *)malloc(mem_size_b);
    float *h_c = (float *)malloc(mem_size_c);
    if (h_a == NULL || h_b == NULL || h_c == NULL) {
        printf("Fail to malloc.\n");
        return 1;
    }

    // Initialize 
    srand(0);
    GenMatrix(height_a, width_a, h_a);
    GenMatrix(height_b, width_b, h_b);

    // CPU
    time_t t = clock();
    GemmHostV1(height_a, width_b, width_a, h_a, width_a,h_b, width_b, h_c, width_b);
    printf("cpu version 1 -> time: %f s, mean value = %f\n", double(clock() - t)/CLOCKS_PER_SEC, GetMean(h_c, height_a, width_b));
    //MatrixPrint(h_c, height_a, width_b);

    t = clock();
    GemmHostV2(height_a, width_b, width_a, h_a, width_a, h_b, width_b, h_c, width_b);
    printf("cpu version 2 -> time: %f s, mean value = %f\n", double(clock() - t)/CLOCKS_PER_SEC, GetMean(h_c, height_a, width_b));
    //MatrixPrint(h_c, height_a, width_b);

    // GPU
    // Allocate memory in host. 
    float msec_total;
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc((void **)&d_a, mem_size_a));
    CUDA_CHECK(hipMalloc((void **)&d_b, mem_size_b));
    CUDA_CHECK(hipMalloc((void **)&d_c, mem_size_c));

    TEST_CUDA_MODULE_UKERNEL(1);
    TEST_CUDA_MODULE_UKERNEL(2);
    TEST_CUDA_MODULE_UKERNEL(3);

    // GPU Device 0: "Tesla T4" with compute capability 7.5 with 40 multi-processors.

    // cpu version 1 -> time: 352.808640 s, mean value = 4721666173127589101568.000000
    // cpu version 2 -> time: 252.558702 s, mean value = 4721666173127589101568.000000
    // gpu version 1 -> time: 0.035052 s, mean value = 4721666173127589101568.000000
    // gpu version 2 -> time: 0.027406 s, mean value = 4721666173127589101568.000000
    // gpu version 3 -> time: 0.013027 s, mean value = 4721666173127589101568.000000
    
    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    CleanUpEnvironment();

    return 0;
}
